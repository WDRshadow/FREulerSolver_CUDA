#include "hip/hip_runtime.h"
#include "limiter.h"
#include "euler_eq.cuh"

struct Matrix4d
{
    double J[4][4];

    __device__ Vec4 operator*(const Vec4 &U) const
    {
        return {
            J[0][0] * U.x + J[0][1] * U.y + J[0][2] * U.z + J[0][3] * U.w,
            J[1][0] * U.x + J[1][1] * U.y + J[1][2] * U.z + J[1][3] * U.w,
            J[2][0] * U.x + J[2][1] * U.y + J[2][2] * U.z + J[2][3] * U.w,
            J[3][0] * U.x + J[3][1] * U.y + J[3][2] * U.z + J[3][3] * U.w};
    }
};

__device__ double minmod(double a, double b, double c)
{
    if (a * b <= 0 || a * c <= 0 || b * c <= 0)
    {
        return 0.0;
    }
    return fmin(fabs(a), fmin(fabs(b), fabs(c))) * (a > 0 ? 1 : -1);
}

__device__ double tvb(double a, double b, double c, double Mh2)
{
    if (fabs(a) <= Mh2)
        return a;
    return minmod(a, b, c);
}

__device__ void eigenDecomposeX(const Vec4 &U, Matrix4d &RX, Matrix4d &RinvX, const double gamma)
{
    const double c = sqrt(gamma * _p / _rho);
    const double H = (_e + _p) / _rho;
    const double u = _u;
    const double v = _v;
    const double ke = 0.5 * (u * u + v * v);

    RX.J[0][0] = 1.0;
    RX.J[1][0] = u - c;
    RX.J[2][0] = v;
    RX.J[3][0] = H - u * c;

    RX.J[0][1] = 1.0;
    RX.J[1][1] = u;
    RX.J[2][1] = v;
    RX.J[3][1] = ke;

    RX.J[0][2] = 0.0;
    RX.J[1][2] = 0.0;
    RX.J[2][2] = 1.0;
    RX.J[3][2] = v;

    RX.J[0][3] = 1.0;
    RX.J[1][3] = u + c;
    RX.J[2][3] = v;
    RX.J[3][3] = H + u * c;

    const double inv_H_ke = 1.0 / (H - ke);
    const double inv_c = 1.0 / c;

    RinvX.J[0][0] = (H * u - c * ke + c * (u * u + v * v) - ke * u) * 0.5 * inv_c * inv_H_ke;
    RinvX.J[0][1] = (-H - c * u + ke) * 0.5 * inv_c * inv_H_ke;
    RinvX.J[0][2] = -v * 0.5 * inv_H_ke;
    RinvX.J[0][3] = 0.5 * inv_H_ke;

    RinvX.J[1][0] = (H - (u * u + v * v)) * inv_H_ke;
    RinvX.J[1][1] = u * inv_H_ke;
    RinvX.J[1][2] = v * inv_H_ke;
    RinvX.J[1][3] = -inv_H_ke;

    RinvX.J[2][0] = -v;
    RinvX.J[2][1] = 0.0;
    RinvX.J[2][2] = 1.0;
    RinvX.J[2][3] = 0.0;

    RinvX.J[3][0] = (-H * u - c * ke + c * (u * u + v * v) + ke * u) * 0.5 * inv_c * inv_H_ke;
    RinvX.J[3][1] = (H - c * u - ke) * 0.5 * inv_c * inv_H_ke;
    RinvX.J[3][2] = -v * 0.5 * inv_H_ke;
    RinvX.J[3][3] = 0.5 * inv_H_ke;
}

__device__ void eigenDecomposeY(const Vec4 &U, Matrix4d &RY, Matrix4d &RinvY, const double gamma)
{
    const double c = sqrt(gamma * _p / _rho);
    const double H = (_e + _p) / _rho;
    const double u = _u;
    const double v = _v;
    const double ke = 0.5 * (u * u + v * v);

    RY.J[0][0] = 1.0;
    RY.J[1][0] = u;
    RY.J[2][0] = v - c;
    RY.J[3][0] = H - v * c;

    RY.J[0][1] = 1.0;
    RY.J[1][1] = u;
    RY.J[2][1] = v;
    RY.J[3][1] = ke;

    RY.J[0][2] = 0.0;
    RY.J[1][2] = -1.0;
    RY.J[2][2] = 0.0;
    RY.J[3][2] = -u;

    RY.J[0][3] = 1.0;
    RY.J[1][3] = u;
    RY.J[2][3] = v + c;
    RY.J[3][3] = H + v * c;

    const double inv_H_ke = 1.0 / (H - ke);
    const double inv_c = 1.0 / c;

    RinvY.J[0][0] = (H * v - c * ke + c * (u * u + v * v) - ke * v) * 0.5 * inv_c * inv_H_ke;
    RinvY.J[0][1] = -u * 0.5 * inv_H_ke;
    RinvY.J[0][2] = (-H - c * v + ke) * 0.5 * inv_c * inv_H_ke;
    RinvY.J[0][3] = 0.5 * inv_H_ke;

    RinvY.J[1][0] = (H - (u * u + v * v)) * inv_H_ke;
    RinvY.J[1][1] = u * inv_H_ke;
    RinvY.J[1][2] = v * inv_H_ke;
    RinvY.J[1][3] = -inv_H_ke;

    RinvY.J[2][0] = u;
    RinvY.J[2][1] = -1.0;
    RinvY.J[2][2] = 0.0;
    RinvY.J[2][3] = 0.0;

    RinvY.J[3][0] = (-H * v - c * ke + c * (u * u + v * v) + ke * v) * 0.5 * inv_c * inv_H_ke;
    RinvY.J[3][1] = -u * 0.5 * inv_H_ke;
    RinvY.J[3][2] = (H - c * v - ke) * 0.5 * inv_c * inv_H_ke;
    RinvY.J[3][3] = 0.5 * inv_H_ke;
}

__device__ __forceinline__ Vec4 cell_mean(const Vec4 *nodes)
{
    constexpr double gll_weight_2d[] = {
        1.0 / 36.0,
        1.0 / 9.0,
        1.0 / 36.0,
        1.0 / 9.0,
        1.0 / 36.0,
        1.0 / 9.0,
        1.0 / 36.0,
        1.0 / 9.0,
        4.0 / 9.0};
    Vec4 mean{};
    for (int i = 0; i < 9; ++i)
    {
        mean += nodes[i] * gll_weight_2d[i];
    }
    return mean;
}

__device__ __forceinline__ Vec4 face_mean(const Vec4 &left, const Vec4 &center, const Vec4 &right)
{
    return left / 6.0 + center * 2.0 / 3.0 + right / 6.0;
}

__device__ void limiter_1d(
    const Vec4 &left_mean,
    const Vec4 &local_mean,
    const Vec4 &right_mean,
    const Vec4 &local_left,
    const Vec4 &local_right,
    const Matrix4d &R,
    const Matrix4d &Rinv,
    Vec4 &new_local_left,
    Vec4 &new_local_right,
    const double h,
    const double Mh2)
{
    const Vec4 left_mean_V = Rinv * left_mean;
    const Vec4 local_mean_V = Rinv * local_mean;
    const Vec4 right_mean_V = Rinv * right_mean;
    const Vec4 local_left_V = Rinv * local_left;
    const Vec4 local_right_V = Rinv * local_right;

    const Vec4 delta_V = (local_right_V - local_left_V) / h;
    const Vec4 D_left_V = (local_mean_V - left_mean_V) / h;
    const Vec4 D_right_V = (right_mean_V - local_mean_V) / h;

    Vec4 delta_mod_V;
    for (int i = 0; i < 4; ++i)
    {
        delta_mod_V[i] = tvb(delta_V[i], D_left_V[i], D_right_V[i], Mh2);
    }
    const Vec4 local_left_mod = R * (local_mean_V - delta_mod_V * 0.5 * h);
    const Vec4 local_right_mod = R * (local_mean_V + delta_mod_V * 0.5 * h);
    new_local_left += local_left_mod * 0.5;
    new_local_right += local_right_mod * 0.5;
}

__device__ __forceinline__ int idx(const int i, const int j)
{
    const int id = j * 3 + i;
    constexpr int _idx[] = {0, 1, 2, 7, 8, 3, 6, 5, 4};
    return _idx[id];
}

__device__ __forceinline__ void set_bc(const Vec4 &bc, const Vec4 *local_U, const int bc_type, Vec4 *neighbour_U,
                                       const double gamma)
{
    for (int j = 0; j < 3; ++j)
    {
        if (bc_type == X_WALL)
        {
            neighbour_U[j] = Vec4(local_U[j][0], -local_U[j][1], local_U[j][2], local_U[j][3]);
        }
        else if (bc_type == Y_WALL)
        {
            neighbour_U[j] = Vec4(local_U[j][0], local_U[j][1], -local_U[j][2], local_U[j][3]);
        }
        else if (bc_type == INLET)
        {
            neighbour_U[j] = bc;
        }
        else
        {
            neighbour_U[j] = local_U[j];
        }
    }
}

__global__ void limiter_kernel(
    const Cell *d_cells,
    const Face *d_faces,
    const Vec4 *d_nodes,
    Vec4 *d_new_nodes,
    const int num_elements,
    const Vec4 bc_Q,
    const double gamma,
    const double hx,
    const double hy,
    const double Mh2)
{
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num_elements)
    {
        return;
    }
    const Cell &cell = d_cells[i];
    if (!cell.isValid)
    {
        return;
    }
    const Vec4 *node = d_nodes + i * 9;
    Vec4 *new_node = d_new_nodes + i * 9;
    for (int j = 0; j < 9; ++j)
    {
        new_node[j] = Vec4();
    }
    const Vec4 U_mean = cell_mean(node);
    Matrix4d RX, RinvX, RY, RinvY;
    eigenDecomposeX(U_mean, RX, RinvX, gamma);
    eigenDecomposeY(U_mean, RY, RinvY, gamma);

    Vec4 local_mean_X[3], local_mean_Y[3];
    for (int j = 0; j < 3; ++j)
    {
        local_mean_X[j] = face_mean(node[idx(0, j)], node[idx(1, j)], node[idx(2, j)]);
        local_mean_Y[j] = face_mean(node[idx(j, 0)], node[idx(j, 1)], node[idx(j, 2)]);
    }

    Vec4 left_mean[3], right_mean[3], bottom_mean[3], top_mean[3];

    const Face &bottom_face = d_faces[cell.faceIds[0]];
    const Face &right_face = d_faces[cell.faceIds[1]];
    const Face &top_face = d_faces[cell.faceIds[2]];
    const Face &left_face = d_faces[cell.faceIds[3]];

    // bottom cell
    if (bottom_face.leftCell < 0)
    {
        set_bc(bc_Q, local_mean_Y, bottom_face.leftCell, bottom_mean, gamma);
    }
    else
    {
        const Vec4 *bottom_node = d_nodes + bottom_face.leftCell * 9;
        for (int j = 0; j < 3; ++j)
        {
            bottom_mean[j] = face_mean(bottom_node[idx(j, 0)], bottom_node[idx(j, 1)], bottom_node[idx(j, 2)]);
        }
    }

    // right cell
    if (right_face.rightCell < 0)
    {
        set_bc(bc_Q, local_mean_X, right_face.rightCell, right_mean, gamma);
    }
    else
    {
        const Vec4 *right_node = d_nodes + right_face.rightCell * 9;
        for (int j = 0; j < 3; ++j)
        {
            right_mean[j] = face_mean(right_node[idx(0, j)], right_node[idx(1, j)], right_node[idx(2, j)]);
        }
    }

    // top cell
    if (top_face.rightCell < 0)
    {
        set_bc(bc_Q, local_mean_Y, top_face.rightCell, top_mean, gamma);
    }
    else
    {
        const Vec4 *top_node = d_nodes + top_face.rightCell * 9;
        for (int j = 0; j < 3; ++j)
        {
            top_mean[j] = face_mean(top_node[idx(j, 0)], top_node[idx(j, 1)], top_node[idx(j, 2)]);
        }
    }

    // left cell
    if (left_face.leftCell < 0)
    {
        set_bc(bc_Q, local_mean_X, left_face.leftCell, left_mean, gamma);
    }
    else
    {
        const Vec4 *left_node = d_nodes + left_face.leftCell * 9;
        for (int j = 0; j < 3; ++j)
        {
            left_mean[j] = face_mean(left_node[idx(0, j)], left_node[idx(1, j)], left_node[idx(2, j)]);
        }
    }

    for (int j = 0; j < 3; ++j)
    {
        // X
        new_node[idx(1, j)] += local_mean_X[j] * 0.5;
        limiter_1d(
            left_mean[j],
            local_mean_X[j],
            right_mean[j],
            node[idx(0, j)],
            node[idx(2, j)],
            RX,
            RinvX,
            new_node[idx(0, j)],
            new_node[idx(2, j)],
            hx,
            Mh2);
        // Y
        new_node[idx(j, 1)] += local_mean_Y[j] * 0.5;
        limiter_1d(
            bottom_mean[j],
            local_mean_Y[j],
            top_mean[j],
            node[idx(j, 0)],
            node[idx(j, 2)],
            RY,
            RinvY,
            new_node[idx(j, 0)],
            new_node[idx(j, 2)],
            hy,
            Mh2);
    }
}

void tvd_limiter(
    const Cell *d_cells,
    const Face *d_faces,
    const Vec4 *d_nodes,
    Vec4 *d_new_nodes,
    const int num_elements,
    const Vec4 &bc_Q,
    const double gamma,
    const double hx,
    const double hy,
    const double Mh2)
{
    const int num_blocks = (num_elements + BLOCK_SIZE - 1) / BLOCK_SIZE;
    limiter_kernel<<<num_blocks, BLOCK_SIZE>>>(
        d_cells, d_faces, d_nodes, d_new_nodes, num_elements, bc_Q, gamma, hx, hy, Mh2);
    hipDeviceSynchronize();
}
